
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void loop(int n) {
	printf("T%d/B%d = %d\n", threadIdx.x, blockIdx.x, n);
}

int main(int argc, char* argv[]) {
	int n;

	hipDeviceSynchronize();

	n = -1;
	if (argc == 2) n = atoi(argv[1]); printf("n = %d\n", n);
	loop<<<2, 2>>>(n);

	hipDeviceSynchronize();

	n = -1;
	printf("n = "); fflush(stdout); scanf("%d", &n);
	loop<<<2, 2>>>(n);

	hipDeviceSynchronize();

	return 0;
}
